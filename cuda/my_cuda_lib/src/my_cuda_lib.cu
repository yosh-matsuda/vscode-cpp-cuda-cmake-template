#include "hip/hip_runtime.h"
#include <cstdio>
#include "my_cuda_lib.h"

__global__ void CudaLibFuncImpl() { printf("Hello my CUDA library\n"); }

void CudaLibFunc()
{
    CudaLibFuncImpl<<<1, 1>>>();
    hipDeviceSynchronize();
}
